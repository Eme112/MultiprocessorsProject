#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

uint32_t rows_a, cols_a, rows_b, cols_b, rows_c, cols_c;
uint32_t size_a, size_b, size_c;
double* A = NULL;
double* B = NULL;
double* C = NULL;

int validateSize(uint32_t rows, uint32_t cols, char matrix) {
    FILE* file;
    double value;
    int count = 0;

    if (matrix == 'A')
        file = fopen("matrizA.txt", "r");
    else
        file = fopen("matrizB.txt", "r");

    if (file == NULL) {
        printf("Failed to open the file.\n");
        return 0;
    }

    while (fscanf(file, "%lf", &value) != EOF) {
        count++;
        char nextChar = fgetc(file);
    }

    fclose(file);

    if (count == rows * cols) {
        return count;
    }
    else {
        printf("Invalid size for matrix %c\n", matrix);
        return 0;
    }
}

void askForInputsA() {
    printf("Enter number of rows for matrix A: ");
    scanf("%u", &rows_a);
    printf("Enter number of columns for matrix A: ");
    scanf("%u", &cols_a);
}

void askForInputsB() {
    printf("Enter number of rows for matrix B: ");
    scanf("%u", &rows_b);
    printf("Enter number of columns for matrix B: ");
    scanf("%u", &cols_b);
}

uint8_t validateInputs() {
    if (cols_a == rows_b) {
        // printf("Valid multiplication\n");
        return 1;
    }
    else {
        printf("Multiplication not valid\n");
        return 0;
    }
}

int readMatrixFromFile(const char* filename, double** matrix, uint32_t rows, uint32_t cols) {
    FILE* inputFile = fopen(filename, "r");
    if (inputFile != NULL) {
        *matrix = (double*)malloc(rows * cols * sizeof(double));
        for (uint32_t i = 0; i < rows; i++) {
            for (uint32_t j = 0; j < cols; j++) {
                if (fscanf(inputFile, "%lf", &(*matrix)[i * cols + j]) != 1) {
                    fclose(inputFile);
                    free(*matrix);
                    return 0;
                }
            }
        }
        fclose(inputFile);
        return 1;
    }
    return 0;
}

void resetMatrix(double* C, uint32_t rows, uint32_t cols) {
    for (uint32_t i = 0; i < rows; i++) {
        for (uint32_t j = 0; j < cols; j++) {
            C[i * cols + j] = 0;
        }
    }
}

void multiplyMatrices(double* A, double* B, double* C, uint32_t rows_a, uint32_t cols_a, uint32_t cols_b) {
    for (uint32_t i = 0; i < rows_a; i++) {
        for (uint32_t j = 0; j < cols_b; j++) {
            C[i * cols_b + j] = 0;
            for (uint32_t k = 0; k < cols_a; k++) {
                C[i * cols_b + j] += A[i * cols_a + k] * B[k * cols_b + j];
            }
        }
    }
}

void multiplyMatrices_openmp(double* A, double* B, double* C, uint32_t rows_a, uint32_t cols_a, uint32_t cols_b) {
    omp_set_num_threads(omp_get_num_procs());
#pragma omp parallel for collapse(2)
    for (uint32_t i = 0; i < rows_a; i++) {
        for (uint32_t j = 0; j < cols_b; j++) {
            C[i * cols_b + j] = 0;
            for (uint32_t k = 0; k < cols_a; k++) {
                C[i * cols_b + j] += A[i * cols_a + k] * B[k * cols_b + j];
            }
        }
    }
}

__global__ void multiplyMatrices_cuda_kernel(double* d_A, double* d_B, double* d_C, uint32_t rows_a, uint32_t cols_a, uint32_t cols_b)
{
    // Multiply matrices A and B and store the result in C
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows_a && j < cols_b) {
        d_C[i * cols_b + j] = 0;
        for (uint32_t k = 0; k < cols_a; k++) {
            d_C[i * cols_b + j] += d_A[i * cols_a + k] * d_B[k * cols_b + j];
        }
    }
}

void multiplyMatrices_cuda(double* h_C, double* d_A, double* d_B, double* d_C, uint32_t rows_a, uint32_t cols_a, uint32_t cols_b)
{
    // Launch kernel to multiply A and B
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((rows_a + threadsPerBlock.x - 1) / threadsPerBlock.x, (cols_b + threadsPerBlock.y - 1) / threadsPerBlock.y);
    multiplyMatrices_cuda_kernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, rows_a, cols_a, cols_b);
    // Sync
	hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, size_c * sizeof(double), hipMemcpyDeviceToHost);
}

int compareMatrices(double* C, const char* filename, uint32_t rows, uint32_t cols) {
    FILE* inputFile = fopen(filename, "r");
    if (inputFile != NULL) {
        for (uint32_t i = 0; i < rows; i++) {
            for (uint32_t j = 0; j < cols; j++) {
                double value;
                if (fscanf(inputFile, "%lf", &value) != 1) {
                    fclose(inputFile);
                    return 0;
                }
                if (abs((C[i * cols + j] - value)) > 1e-10) {
                    fclose(inputFile);
                    return 0;
                }
            }
        }
        fclose(inputFile);
        return 1;
    }
    return 0;
}

void print_table(double serial[5], double openmp[5], double cuda[5]) {
    int rows = 8;
    int columns = 4;

    // Print the table header
    printf("| %-12s ", "Corrida ");
    printf("| %-12s ", "Serial");
    printf("| %-12s ", "OpenMP");
    printf("| %-12s ", "CUDA");
    printf("|\n");

    // Print the horizontal line
    for (int j = 0; j < columns; j++) {
        printf("+--------------");
    }
    printf("+\n");

    // Print the table rows
    double promedio_serial = 0;
    double promedio_openmp = 0;
    double promedio_cuda = 0;
    for (int i = 0;i < 5;i++) {
        printf("| %-12d ", i + 1);
        printf("| %-12f ", serial[i]);
        printf("| %-12f ", openmp[i]);
        printf("| %-12f ", cuda[i]);
        printf("|\n");
        promedio_serial += serial[i];
        promedio_openmp += openmp[i];
        promedio_cuda += cuda[i];
    }

    promedio_serial = promedio_serial / 5;
    promedio_openmp = promedio_openmp / 5;
    promedio_cuda = promedio_cuda / 5;
    //print promedios
    printf("| %-12s ", "Promedio");
    printf("| %-12f ", promedio_serial);
    printf("| %-12f ", promedio_openmp);
    printf("| %-12f ", promedio_cuda);
    printf("|\n");
    //print porcentajes
    printf("| %-12s ", "% vs Serial");
    printf("| %-12d ", 100);
    printf("| %-12f ", promedio_openmp / promedio_serial * 100);
    printf("| %-12f ", promedio_cuda / promedio_serial * 100);
    printf("|\n");

    // Print the horizontal line
    for (int j = 0; j < columns; j++) {
        printf("+--------------");
    }
    printf("+\n");

    // Check for fastest method
    if(promedio_serial < promedio_openmp && promedio_serial < promedio_cuda) {
        printf("El método más rápido para realizar la multiplicación de matrices es: SERIAL\n");
    } else if(promedio_openmp < promedio_serial && promedio_openmp < promedio_cuda) {
        printf("El método más rápido para realizar la multiplicación de matrices es: OPENMP\n");
    } else {
        printf("El método más rápido para realizar la multiplicación de matrices es: CUDA\n");
    }
    
}

int main() {
    clock_t start, end;
    double cpu_time_used;
    double serial[5];
    double openmp[5];
    double cuda[5];

    askForInputsA();
    size_a = validateSize(rows_a, cols_a, 'A');
    if (size_a == 0)
        return 0;
    askForInputsB();
    size_b = validateSize(rows_b, cols_b, 'B');
    if (size_b == 0)
        return 0;
    if (!validateInputs())
        return 0;

    // Validate there is enough resources for CUDA with the desired size of matrices
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if( (rows_a > prop.maxThreadsDim[0]) || (cols_b > prop.maxThreadsDim[1])) {
        printf("Error: The size of the matrices is too big for CUDA using the current configuration\n");
        return 0;
    }

    rows_c = rows_a;
    cols_c = cols_b;
    size_c = rows_c * cols_c;

    A = (double*)malloc(size_a * sizeof(double));
    B = (double*)malloc(size_b * sizeof(double));
    C = (double*)malloc(size_c * sizeof(double));

    // hipMallocManaged(&A, size_a * sizeof(double));
    // hipMallocManaged(&B, size_b * sizeof(double));
    // hipMallocManaged(&C, size_c * sizeof(double));

    if (A == NULL || B == NULL || C == NULL) {
        printf("Not enough memory for the desired sizes of matrices\n");
        return 0;
    }

    if (!readMatrixFromFile("matrizA.txt", &A, rows_a, cols_a)) {
        printf("Failed to read matrix A from file.\n");
        free(A);
        free(B);
        free(C);
        return 0;
    }

    if (!readMatrixFromFile("matrizB.txt", &B, rows_b, cols_b)) {
        printf("Failed to read matrix B from file.\n");
        free(A);
        free(B);
        free(C);
        return 0;
    }

    // Start serial multiplication
    for (int i = 0; i < 5; i++) {
        start = clock();
        multiplyMatrices(A, B, C, rows_a, cols_a, cols_b);
        end = clock();
        cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
        serial[i] = cpu_time_used;
    }

    // printf("Serial multiplications completed.\n");

    // Write matrix C to file
    FILE* outputFile = fopen("matrizC.txt", "w");
    if (outputFile != NULL) {
        for (uint32_t i = 0; i < rows_c; i++) {
            for (uint32_t j = 0; j < cols_c; j++) {
                fprintf(outputFile, "%.10f\n", C[i * cols_c + j]);
            }
        }
        fclose(outputFile);
        // printf("Matrix C written to matrizC.txt.\n");
    }
    else {
        printf("Failed to open matrizC.txt for writing.\n");
    }

    // Reset matrix C before executing openmp
    resetMatrix(C, rows_c, cols_c);

    // Start openmp
    for (int i = 0; i < 5; i++) {
        start = clock();
        multiplyMatrices_openmp(A, B, C, rows_a, cols_a, cols_b);
        end = clock();
        cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
        openmp[i] = cpu_time_used;
    }
    // printf("Openmp multiplications completed.\n");

    if (compareMatrices(C, "matrizC.txt", rows_c, cols_c)) {
        printf("OpenMP matches with serial method.\n");
    }
    else {
        printf("OpenMP does not match with serial method.\n");
    }

    // Reset matrix C before executing cuda
    resetMatrix(C, rows_c, cols_c);

    // Start cuda
    double* d_A = NULL;
    double* d_B = NULL;
    double* d_C = NULL;

    // Memory allocation for device copies of A, B, C
    hipMalloc((void**)&d_A, size_a * sizeof(double));
    hipMalloc((void**)&d_B, size_b * sizeof(double));
    hipMalloc((void**)&d_C, size_c * sizeof(double));

    // Copy inputs to device
    hipMemcpy(d_A, A, size_a * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_b * sizeof(double), hipMemcpyHostToDevice);

    for (int i = 0; i < 5; i++) {
        start = clock();
        multiplyMatrices_cuda(C, d_A, d_B, d_C, rows_a, cols_a, cols_b);
        end = clock();
        cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
        cuda[i] = cpu_time_used;
    }
    // printf("Cuda multiplications completed.\n");

    if (compareMatrices(C, "matrizC.txt", rows_c, cols_c)) {
        printf("CUDA matches with serial method.\n");
    }
    else {
        printf("CUDA does not match with serial method.\n");
    }


    print_table(serial, openmp, cuda);
    // Free memory
    free(A);
    free(B);
    free(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}